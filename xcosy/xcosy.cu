#include "hip/hip_runtime.h"
#include"xcosy.h"
#include<hip/hip_runtime.h>
#include<string.h>
#include<string>
#include<fstream>
#include<iostream>
using namespace std;
#define BLOCK_DIM 1024
__global__
void xcosy(int lengthOfX,int lengthOfCon,float *x,float *y,float *con)
{
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	if(i>=lengthOfCon) return;
	int offset=i*lengthOfX;
	float xy=0;
	float xx=0;
	float yy=0;
	for(int j=0;j<lengthOfX;j++){
		xy+=x[j]*y[j+offset];
		xx+=x[j]*x[j];
		yy+=y[j+offset]*y[j+offset];
	}
	con[i]=xy/(sqrt(xx)*sqrt(yy));
}
extern "C" void c_xcosy(int lengthOfX,int lengthOfCon,float *x,float *y,float *con)
{
	int nblocks=(lengthOfCon+BLOCK_DIM-1)/BLOCK_DIM;
	cout<<"using blocks="<<nblocks<<",and the thread is :"<<lengthOfCon<<endl;
	cout<<"start to exec xcos y"<<endl;
	xcosy<<<nblocks,BLOCK_DIM>>>(lengthOfX,lengthOfCon,x,y,con);
	cout<<"end of exec xcosy"<<endl;
	
}
